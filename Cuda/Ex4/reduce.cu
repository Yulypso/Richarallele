
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>


__global__ void reduce_kernel(float *in, float *out)
{
    // TODO : coder ici
}

__host__ void init_vec(float *h_in, int ntot)
{
    for(int i = 0 ; i < ntot ; i++)
    {
	h_in[i] = sinf(float(i));
    }
}

__host__ void verif(float sum, float *h_in, int ntot)
{
    float sum_res = 0.;
    for(int i = 0 ; i < ntot ; i++)
    {
			sum_res += h_in[i];
    }
    float err = fabsf((sum - sum_res)/sum);
    printf("GPU sum : %.4e\n", sum);
    printf("CPU sum : %.4e\n", sum_res);
    if (err < 1.e-4)
    {
			printf("TEST PASSED (err %.4e < 1.e-4).\n", err);
    }
 	  else
    {
			printf("TEST FAILED (err %.4e > 1.e-4).\n", err);
    }
}

int main(int argc, char **argv)
{
    float sum;
    int nthreads, nblocks, ntot;

    nthreads = 128;
    ntot = atoi(argv[1]);
    nblocks = (ntot + nthreads - 1) / nthreads;

    printf("Ntot     : %d\n", ntot);
    printf("nthreads : %d\n", nthreads);
    printf("nblocks  : %d\n", nblocks);

    float *d_sum, *d_bl, *d_in, *h_in;

    h_in = (float*)malloc(ntot*sizeof(float));

    hipMalloc((void**)&d_sum, sizeof(float));
    hipMalloc((void**)&d_bl, nblocks*sizeof(float));
    hipMalloc((void**)&d_in, ntot*sizeof(float));

    init_vec(h_in, ntot);
    hipMemcpy(d_in, h_in, ntot*sizeof(float), hipMemcpyHostToDevice);

    // TODO : la réduction de d_in a lieu ici, le resultat est obtenu dans *d_sum

    hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    
    verif(sum, h_in, ntot);

    hipFree(d_sum);
    hipFree(d_bl);
    hipFree(d_in);
    free(h_in);

    return 0;
}

